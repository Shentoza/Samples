#include "hip/hip_runtime.h"
/*
 * Copyright 1993-2015 NVIDIA Corporation.  All rights reserved.
 *
 * Please refer to the NVIDIA end user license agreement (EULA) associated
 * with this source code for terms and conditions that govern your use of
 * this software. Any use, reproduction, disclosure, or distribution of
 * this software and related documentation outside the terms of the EULA
 * is strictly prohibited.
 *
 */

// This sample is an implementation of a simple line-of-sight algorithm:
// Given a height map and a ray originating at some observation point,
// it computes all the points along the ray that are visible from the
// observation point.
// It is based on the description made in "Guy E. Blelloch.  Vector models
// for data-parallel computing. MIT Press, 1990" and uses open source CUDA
// Thrust Library

#ifdef _WIN32
#  define NOMINMAX
#endif

//First some glew
#include "GL\glew.h"
#include "GL\freeglut.h"


// includes, system
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>
#include <float.h>

// includes, project"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <>
#include <helper_functions.h>
#include <hip/hip_runtime_api.h>
#include <hip/hip_vector_types.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>

// includes, library
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <Thrust\execution_policy.h>
#include <thrust/scan.h>
#include <thrust/copy.h>
#include <cuda_gl_interop.h>


//includes migrated project
//#define TINYOBJLOADER_IMPLEMENTATION

#include "glm\glm.hpp"
#include "glm\gtc\matrix_transform.hpp"
#include "glm/gtx/quaternion.hpp"
#include <iostream>
#include "Helper.h"
#include "InputManager.h"
#include "Transform.h"
#include "Core\Shader_Manager.h"
#include "Core\GameModels.h"
#include "Structs.h"

using namespace glm;

////////////////////////////////////////////////////////////////////////////////
// defines, declaration, variables

#define Samples 1
#define NUM_SPHERES 10
int maxFrames = -1;

// Height field texture reference
Managers::Shader_Manager* shaderManager;
GLuint program;
int uniformLocs[4];

//Geometry
Sphere *hSpheres;
Sphere *dSpheres;

vec2 resolution(1024, 1024);
vec3* cudaAccumulatedBuffer;


//openGL Interop
static hipGraphicsResource* cuda_PixelBufferObject;
GLuint cuda_pixelBuffer;

GLuint host_pixelBuffer;
vec3* hostAccumulatedBuffer;

int frames = 0;
////////////////////////////////////////////////////////////////////////////////
// declaration, forward

//migrated
void renderScene(void);
void closeCallback();
void InitGL(int* argc, char** argv);
void keyboard(unsigned char key, int /*x*/, int /*y*/);
void mouse(int button, int state, int x, int y);
void motion(int x, int y);
void Update();
void createOutputVBO();
__global__ void render_kernel(uchar4* deviceMem, vec3* buffer, int frameNumber, unsigned int frameHash, vec2 resolution, cudaMat4 inverse, Sphere* spheres);
void render_cpu(uchar4* deviceMem, vec3* buffer, int frameNumber, unsigned int frameHash, vec2 resolution, cudaMat4 inverse, Sphere* spheres);
void initCudaAndScene(int* argc, char** argv);

////////////////////////////////////////////////////////////////////////////////
// Program main
////////////////////////////////////////////////////////////////////////////////


int main(int argc, char **argv)
{
	
	InitGL(&argc, argv);
	initCudaAndScene(&argc, argv);
	glutMainLoop();

	checkCudaErrors(hipFree(cudaAccumulatedBuffer));
	checkCudaErrors(hipFree(dSpheres));
	checkCudaErrors(hipHostFree(hSpheres));
	free(hostAccumulatedBuffer);

	delete shaderManager;
	return 0;

}

cudaMat4 inverseWvp;

void renderScene(void)
{
	mat4 wvp = mat4();

	Transform* cameraTransform = InputManager::Instance()->getTransform();
	vec3 camPos = cameraTransform->getPosition();
	vec3 camForward = cameraTransform->getForwardVector();
	vec3 camUp = cameraTransform->getUpVector();
	mat4 rotation = mat4_cast(cameraTransform->getRotation());
	mat4 lookAt = glm::lookAt(camPos, camPos + camForward, camUp);
	mat4 perspective = glm::perspective(3.141592f / 4.0f, resolution.x / resolution.y, 0.1f, 30.0f);
	wvp = perspective * lookAt;
	


	// if we are pathTracing
	if (InputManager::Instance()->pathTracing){
		glClearColor(0.0, 0.0, 0.0, 1.0);
		glClear(GL_COLOR_BUFFER_BIT | GL_DEPTH_BUFFER_BIT);
		

		//Camera Changed, generate new Picture!
		if (InputManager::Instance()->changedCamera){
			InputManager::Instance()->changedCamera = false;
			inverseWvp = glmMat4ToCudaMat4(inverse(wvp));
			frames = 1;
			checkCudaErrors(hipMemset(cudaAccumulatedBuffer, 0, resolution.x * resolution.y * sizeof(vec3)));
			memset(hostAccumulatedBuffer, 0, resolution.x * resolution.y * sizeof(vec3));
		}
		else {
			frames++;
		}
		if (InputManager::Instance()->gpuTracing){
			uchar4* deviceMem;
			size_t num_bytes;
			checkCudaErrors(hipDeviceSynchronize());

			checkCudaErrors(hipGraphicsMapResources(1, &cuda_PixelBufferObject));
			checkCudaErrors(hipGraphicsResourceGetMappedPointer((void**)&deviceMem, &num_bytes, cuda_PixelBufferObject));
		if (maxFrames == -1 || frames <= maxFrames) {


			dim3 blockDim(16, 16);
			dim3 gridDim(resolution.x / 16, resolution.y / 16);
			//int rng =rand();

			render_kernel << < gridDim, blockDim >> >(deviceMem, cudaAccumulatedBuffer, frames, Hash(frames), resolution, inverseWvp, dSpheres);
			//render_kernel << < gridDim, blockDim >> >(deviceMem, cudaAccumulatedBuffer, 0, Hash(rng), resolution, inverseWvp, dSpheres);
			hipError_t errSync = hipGetLastError();
			if (errSync != hipSuccess)
				printf("Sync Kernel error: %s \n", hipGetErrorString(errSync));
			
		}

			checkCudaErrors(hipDeviceSynchronize());
			checkCudaErrors(hipGraphicsUnmapResources(1, &cuda_PixelBufferObject));

			glRasterPos2i(-1, -1);
			glBindBufferARB(GL_PIXEL_UNPACK_BUFFER_ARB, cuda_pixelBuffer);
			glDrawPixels(resolution.x, resolution.y, GL_RGBA, GL_UNSIGNED_BYTE, 0);
			glBindBufferARB(GL_PIXEL_UNPACK_BUFFER_ARB, 0);
			glutSwapBuffers();
		}
		else {
			if (maxFrames == -1 || frames <= maxFrames) {
				uchar4* hostMem = (uchar4*)malloc(sizeof(uchar4) * resolution.x * resolution.y);;
				render_cpu(hostMem, hostAccumulatedBuffer, frames, Hash(frames), resolution, inverseWvp, hSpheres);
				glRasterPos2i(-1, -1);
				glDrawPixels(resolution.x, resolution.y, GL_RGBA, GL_UNSIGNED_BYTE, hostAccumulatedBuffer);
				glutSwapBuffers();
			}
		}
	}


	//openGL Geometry Rendering
	else {
		glClear(GL_COLOR_BUFFER_BIT | GL_DEPTH_BUFFER_BIT);
		glUseProgram(program);
		glUniformMatrix4fv(uniformLocs[0], 1, FALSE, (float*)&wvp);
		glUniformMatrix4fv(uniformLocs[1], 1, FALSE, (float*)&rotation);

		std::map<std::string, Models::Model*>::iterator it;
		Models::GameModels* gameModels = Models::GameModels::Instance();
		for (it = gameModels->GameModelList.begin(); it != gameModels->GameModelList.end(); ++it){
			Models::Model model = *it->second;
			glUniformMatrix4fv(uniformLocs[2], 1, FALSE, (float*)&model.modelTransform->getModelMatrix());
			vec3 col = model.color;
			glUniform3f(uniformLocs[3], col.x, col.y, col.z);
			printOglError("kernel.cu", 223);
			glBindVertexArray(model.vao);
			glDrawArrays(GL_TRIANGLES, 0, model.indicesCount);
			glBindVertexArray(0);
		}
		glUseProgram(0);
		glutSwapBuffers();
	}
}

//Checks if a given Ray has an intersection with an object (currently only spheres)
__device__ __host__ bool hasIntersection(const Ray &r, float &minDistance, int &sphereIndex, OBJECT_TYPE &objectType, Sphere* spheres, int &avoid){


	float distanceToSphere = 1e21;
	float inf = minDistance = 1e21;

	for (int i = 0; i < NUM_SPHERES; ++i) {
		distanceToSphere = spheres[i].intersect(r);
		if (avoid != i && abs(distanceToSphere) > EPSILON && distanceToSphere < minDistance){
			minDistance = distanceToSphere;
			sphereIndex = i;
			objectType = OBJECT_TYPE::SPHERE;
		}
	}

	return minDistance < inf;
}

__device__ __host__ vec3 sampleHemisphere(vec3 d, float phi, float sina, float cosa) {
	vec3 w = normalize(d), u = normalize(cross(vec3(w.y, w.z, w.x), w)), v = cross(w, u);
	return (u*cos(phi) + v*sin(phi)) * sina + w * cosa;
}

//Compute path bounces, accumulate colors
__device__ __host__ vec3 radiance(Ray &r, hiprandState *randstate, Sphere* spheres){


	vec3 mask(1.0f, 1.0f, 1.0f);

	vec3 color(0.0f, 0.0f, 0.0f);
	int avoid = -1;
	for (int bounce = 0; bounce < 8; ++bounce) {
		//Scene Intersections
		float t; //distance to intersection
		int sphereIndex = -1;
		OBJECT_TYPE objectType = OBJECT_TYPE::NONE;
		vec3 primitiveColor;
		vec3 emissionColor;
		vec3 intersection;
		vec3 normal;
		vec3 n1; //oriented normal
		vec3 nextDir;
		refl_t reflType;

		if (!hasIntersection(r, t, sphereIndex, objectType, spheres, avoid))
		{
			return vec3(0.0f);
		}

		//determine the properties of the object hit
		//Currently only spheres
		switch (objectType) {
			case OBJECT_TYPE::SPHERE:
				Sphere &sp = spheres[sphereIndex]; 
				intersection = getPointOnRay(r, t);
				normal = normalize(intersection - sp.pos);
				n1 = normal * sign(-dot(normal, r.dir));
				primitiveColor = sp.colour;
				reflType = sp.refl;
				avoid = sphereIndex;
				emissionColor = sp.emi;
				//Debug: See all the shapes how they are
				//return primitiveColor;
				color += (mask * emissionColor);
				break;
		}


		//different Shading types
		switch (reflType) {
		case DIFF:
		{
			//2 random numbers
			float rn1 = 0.f, rn2 = 0.f;
#ifdef __CUDA_ARCH__
			rn1 = hiprand_uniform(randstate);
			rn2 = hiprand_uniform(randstate);
#else
			rn1 = (float)rand() / RAND_MAX;
			rn2 = (float)rand() / RAND_MAX;
#endif
			//cosine weighted sampling
			nextDir = sampleHemisphere(n1, rn1*TWO_PI, sqrtf(rn2), sqrtf(1. - rn2));
			//intersection += n1 * 0.03f;
			mask *= primitiveColor;
			break;
		}
		case SPEC:
			nextDir = reflect(r.dir, normal);
			//intersection += n1 * 0.03f;
			mask *= primitiveColor;

			break;


			//ideal refraction (smallpt code by Kevin Beason)
		case REFR:
			float a = dot(normal, r.dir), ddn = abs(a);
			float nc = 1., nt = 1.5, nnt = mix(nc / nt, nt / nc, float(a>0.));
			float cos2t = 1. - nnt*nnt*(1. - ddn*ddn);
			nextDir = reflect(r.dir, normal);
			
			if (cos2t>0.) {
				vec3 tdir = normalize(r.dir*nnt + sign(a)*normal*(ddn*nnt + sqrt(cos2t)));
				float R0 = (nt - nc)*(nt - nc) / ((nt + nc)*(nt + nc)),
					c = 1. - mix(ddn, dot(tdir, normal), float(a>0.));
				float Re = R0 + (1. - R0)*c*c*c*c*c, P = .25 + .5*Re, RP = Re / P, TP = (1. - Re) / (1. - P);
				//intersection += 0.01f * n1;
#ifdef __CUDA_ARCH__
				if (hiprand_uniform(randstate) < P) { mask *= RP; }
#else
				if (((float)rand() / RAND_MAX) < P) { mask *= RP; }
#endif
				else { mask *= primitiveColor*TP; nextDir = tdir;}
			}
			break;
		}
		
		r.orig = intersection;
		r.dir = nextDir;
	}
	return color;
}

__global__ void render_kernel(uchar4* deviceMem, vec3* buffer, int frameNumber, unsigned int frameHash, vec2 resolution, cudaMat4 inverse, Sphere* spheres){
	unsigned int pixelX = (blockIdx.x * blockDim.x) + threadIdx.x;
	unsigned int pixelY = (blockIdx.y* blockDim.y) + threadIdx.y;
	
	int blockId = blockIdx.x + blockIdx.y * gridDim.y;
	int threadId = blockId * (blockDim.x * blockDim.y) + (threadIdx.y * blockDim.x) + threadIdx.x;

	int i = (pixelY * resolution.x) + pixelX;

	hiprandState randState;
	hiprand_init(frameHash + threadId, 0, 0, &randState);

	//Convert Pixel Coords to Screenspace
	vec4 nearPoint{ 2.0f * (float)pixelX / resolution.x - 1.0f,  2.0f * (float)pixelY / resolution.y - 1.0f, 0.1f, 1.0f };
	vec4 farPoint { 2.0f * (float)pixelX / resolution.x - 1.0f,  2.0f * (float)pixelY / resolution.y  - 1.0f, 0.9f, 1.0f};
	
	
	nearPoint = multiplyMV(inverse, nearPoint);
	farPoint = multiplyMV(inverse, farPoint);
	nearPoint /= nearPoint.w;
	farPoint /= farPoint.w;

	vec4 rayD = normalize(farPoint - nearPoint);
	Ray ray(vec3(nearPoint.x, nearPoint.y, nearPoint.z), vec3( rayD.x, rayD.y, rayD.z ));



	vec3 finalColor(0, 0, 0);
	for (int s = 0; s < Samples; ++s){
		// Radiance
		finalColor += radiance(ray, &randState, spheres) * (1.0f / (float)Samples);
	}
	buffer[i] += finalColor;
	vec3 tempCol = (buffer[i]/(float)frameNumber);

	
	vec3 colour = vec3(clamp(tempCol.x, 0.0f, 1.0f), clamp(tempCol.y, 0.0f, 1.0f), clamp(tempCol.z, 0.0f, 1.0f));
	//convert from 96-bit to 24-bit colour + perform gamma correction
	deviceMem[i] = make_uchar4((unsigned char)(powf(colour.x, 1 / 2.2f) * 255), (unsigned char)(powf(colour.y, 1/ 2.2f) * 255), (unsigned char)(powf(colour.z, 1.f / 2.2f) * 255.f), 1);
}

void render_cpu(uchar4* mem, vec3* buffer, int frameNumber, unsigned int frameHash, vec2 resolution, cudaMat4 inverse, Sphere* spheres){
	for (int pixelX = 0; pixelX < resolution.x; ++pixelX)
		for (int pixelY = 0; pixelY < resolution.y; ++pixelY) {
			int i = (pixelY * resolution.x) + pixelX;

			srand(frameHash + i);


			vec4 nearPoint{ 2.0f * (float)pixelX / resolution.x - 1.0f, 2.0f * (float)pixelY / resolution.y - 1.0f, 0.1f, 1.0f };
			vec4 farPoint{ 2.0f * (float)pixelX / resolution.x - 1.0f, 2.0f * (float)pixelY / resolution.y - 1.0f, 0.9f, 1.0f };


			nearPoint = multiplyMV(inverse, nearPoint);
			farPoint = multiplyMV(inverse, farPoint);
			nearPoint /= nearPoint.w;
			farPoint /= farPoint.w;

			vec4 rayD = normalize(farPoint - nearPoint);
			Ray ray(vec3(nearPoint.x, nearPoint.y, nearPoint.z), vec3(rayD.x, rayD.y, rayD.z));



			vec3 finalColor(0, 0, 0);
			for (int s = 0; s < Samples; ++s){
				// Radiance
				finalColor += radiance(ray, 0, spheres) * (1.0f / (float)Samples);
			}
			buffer[i] += finalColor;
			vec3 tempCol = (buffer[i] / (float)frameNumber);


			vec3 colour = vec3(clamp(tempCol.x, 0.0f, 1.0f), clamp(tempCol.y, 0.0f, 1.0f), clamp(tempCol.z, 0.0f, 1.0f));
			//convert from 96-bit to 24-bit colour + perform gamma correction
			mem[i] = make_uchar4((unsigned char)(powf(colour.x, 1 / 2.2f) * 255), (unsigned char)(powf(colour.y, 1 / 2.2f) * 255), (unsigned char)(powf(colour.z, 1.f / 2.2f) * 255.f), 1);
		}
}

void closeCallback()
{
	std::cout << "GLUT:\t Finished" << std::endl;
	glutLeaveMainLoop();
}

void resize(int width, int height) {
	glutReshapeWindow(resolution.x, resolution.y);
}

void InitGL(int* argc, char **argv)
{
	glutInit(argc, argv);
	//glutInitDisplayMode(GLUT_DEPTH | GLUT_DOUBLE | GLUT_RGBA);
	glutInitDisplayMode(GLUT_DOUBLE | GLUT_RGBA);
	
	glutInitWindowPosition(600, 0);
	glutInitWindowSize(resolution.x, resolution.y);

	glutCreateWindow("MC Tracer");
	glutSetOption(GLUT_ACTION_ON_WINDOW_CLOSE, GLUT_ACTION_GLUTMAINLOOP_RETURNS);
	glutDisplayFunc(Update);
	glutKeyboardFunc(keyboard);
	glutMouseFunc(mouse);
	glutMotionFunc(motion);
	glutReshapeFunc(resize);
	glutCloseFunc(closeCallback);
	//glutIdleFunc(Update);

	
	glewInit();
	printOglError("Main.cpp", 433);
	if (glewIsSupported("GL_VERSION_4_5")) //lower your version if 4.5 is not supported by your video card
	{
		std::cout << " OpenGL Version is 4.5\n ";
	}
	else
	{
		std::cout << "OpenGL 4.5 not supported\n ";
	}
	printOglError("Main.cpp", 442);
	//Create output VBO
	createOutputVBO();
	setVSync(false);

	glEnable(GL_DEPTH_TEST);
	glCullFace(GL_BACK);
	glClearColor(0.0f, 0.0f, 0.0f, 0.0f);

	printOglError("Main.cpp", 450);
	glFrontFace(GL_CCW);

	//load and compile shaders
	shaderManager = new Managers::Shader_Manager(); //thanks to Erik
	// for pointing this out
	shaderManager->CreateProgram("colorShader",
		"Shaders\\Vertex_Shader.glsl",
		"Shaders\\Fragment_Shader.glsl");
	program = Managers::Shader_Manager::GetShader("colorShader");
	uniformLocs[0] = glGetUniformLocation(program, "wvp");
	uniformLocs[1] = glGetUniformLocation(program, "rotation");
	uniformLocs[2] = glGetUniformLocation(program, "modelMat");
	uniformLocs[3] = glGetUniformLocation(program, "uColor");
}

void initCudaAndScene(int* argc, char** argv){
	if (*argc >= 2) {
		if (sscanf(argv[1], "%i", &maxFrames) != 1) {
			maxFrames = -1;
			fprintf(stderr, "error - not an integer");
		}
		else
			printf("Max Frames: %i", maxFrames);
	}
	//Allocate Memory
	checkCudaErrors(hipMalloc(&cudaAccumulatedBuffer, resolution.x * resolution.y * sizeof(vec3)));
	checkCudaErrors(hipHostAlloc(&hSpheres, NUM_SPHERES * sizeof(Sphere), hipHostMallocDefault));
	checkCudaErrors(hipMalloc(&dSpheres, NUM_SPHERES * sizeof(Sphere)));
	Models::GameModels* gameModels = Models::GameModels::Instance();


	//Write spheres
	//Format::
	//				Radius	Position					Emission				Color							Material
	//LIGHT
	hSpheres[0] = { 4.f,	{ 0, 10, 0 },				{ 12.0f,12.0f,12.0f},		{ 0.0f, 0.0f, 0.0f },	DIFF };
	//Cornell box
	hSpheres[1] = { 1000,	{ 1010, 0, 0 },				{ 0, 0, 0 },			{ 0.75f, 0.25f, 0.25f },		DIFF}; //Left
	hSpheres[2] = { 1000,	{ -1010, 0, 0 },			{ 0, 0, 0 },			{ 0.25f, 0.25f, 0.75f },		DIFF }; //Right
	hSpheres[3] = { 1000,	{ 0, 0, 1010},				{ 0, 0, 0 },			{ 0.f, .75f, .75f },				SPEC}; //Front
	hSpheres[4] = { 1000,	{ 0, 0, -1030 },			{ 0, 0, 0 },			{ 0.75f, 0.2f, 0.2f },			DIFF }; //Back
	hSpheres[5] = { 1000,	{ 0, 1010, 0 },				{ 0, 0, 0 },			{ 0.0f, 0.75f, 0.0f },			DIFF}; //Top
	hSpheres[6] = { 1000,	{ 0, -1010, 0 },			{ 0, 0, 0 },			{ 0.8f, 0.8f, .8f },			DIFF}; //Bottom
	//2 Spheres
	hSpheres[7] = { 3.f,	{ -3.f, -7.f, 4.f },		{ 0, 0, 0 },			{ 1.f, 1.f, 1.f },				SPEC };
	hSpheres[8] = { 2.5f,	{ 3., -4.5f, -2.f },		{ 0, 0, 0 },			{ .7f, 1.f, .9f },				REFR };
	hSpheres[9] = { 1.75f,	{ -3.f, -8.25f, -4.f },		{ 0, 0, 0 },			{ 1.f, 1.f, 0.f },				DIFF};
	for (int i = 0; i < NUM_SPHERES; ++i){
		Sphere s = hSpheres[i];
		vec3 scale = vec3(s.rad);
		vec3 pos = vec3(s.pos.x, s.pos.y, s.pos.z);
		quat rot = quat();
		std::string name = "sphere" + std::to_string(i);
		std::string path = "Resources/Sphere.obj";
		gameModels->LoadObjFile(name, path);
		gameModels->SetTransform(name, scale, pos, rot);
		gameModels->SetColor(name, s.colour);
	}
	checkCudaErrors(hipMemcpy(dSpheres, hSpheres, sizeof(Sphere) * NUM_SPHERES, hipMemcpyHostToDevice));
	checkCudaErrors(hipGraphicsGLRegisterBuffer(&cuda_PixelBufferObject, cuda_pixelBuffer, cudaGraphicsMapFlagsNone));

	hostAccumulatedBuffer = (vec3*)malloc(sizeof(vec3) * resolution.x * resolution.y);
}

int lastTime = 0;
void Update() {
	int currentTime = glutGet(GLUT_ELAPSED_TIME);

	char framerate[100];
	int difference = currentTime - lastTime;
	sprintf(framerate, "FPS: %4.2f", 1000.0f / (float)difference);
	glutSetWindowTitle(framerate);


	renderScene();
	lastTime = currentTime;
}

void keyboard(unsigned char key, int /*x*/, int /*y*/)
{
	if (key == 27){
#if defined(__APPLE__) || defined(MACOSX)
		exit(EXIT_SUCCESS);
#else
		glutDestroyWindow(glutGetWindow());
		return;
#endif
	}
	else {
		InputManager::Instance()->UpdateKeyboard(key);
	}
}

void mouse(int button, int state, int x, int y)
{
	InputManager::Instance()->UpdateMouse(button, state, x, y);
}

void motion(int x, int y)
{
	InputManager::Instance()->UpdateMotion(x, y);
}

void createOutputVBO()
{
	//create vertex buffer object
	glGenBuffersARB(1, &cuda_pixelBuffer);
	glBindBufferARB(GL_PIXEL_UNPACK_BUFFER_ARB, cuda_pixelBuffer);
	glBufferDataARB(GL_PIXEL_UNPACK_BUFFER_ARB, resolution.x * resolution.y * sizeof(uchar4), 0, GL_STREAM_DRAW_ARB);
	glBindBufferARB(GL_PIXEL_UNPACK_BUFFER_ARB, 0);

	/*glGenBuffersARB(1, &host_pixelBuffer);
	glBindBufferARB(GL_PIXEL_UNPACK_BUFFER_ARB, host_pixelBuffer);
	glBufferDataARB(GL_PIXEL_UNPACK_BUFFER_ARB, resolution.x * resolution.y * sizeof(uchar4), 0, GL_STREAM_DRAW_ARB);
	glBindBufferARB(GL_PIXEL_UNPACK_BUFFER_ARB, 0);*/
}